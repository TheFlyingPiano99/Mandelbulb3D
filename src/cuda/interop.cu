#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "interop.cuh"
#include "stdio.h"
#include <memory>
#include <cuda/std/complex>
#include "../glfwim/input_manager.h"
#include "../camera/perspective_camera.h"
#include "../mandelbulb/mandelbulb_animator.h"
#include "glm/gtc/matrix_transform.hpp"

#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

uint32_t imageWidth, imageHeight;
hipExternalMemory_t cudaExtMemImageBuffer; // memory handler to the imported memory allocation
hipMipmappedArray_t cudaMipmappedImageArray; // the image interpreted as a mipmapped array
hipSurfaceObject_t surfaceObject; // surface object to the first mip level of the array. Allows write

void freeExportedVulkanImage()
{
    checkCudaError(hipDestroySurfaceObject(surfaceObject));
    checkCudaError(hipFreeMipmappedArray(cudaMipmappedImageArray));
    checkCudaError(hipDestroyExternalMemory(cudaExtMemImageBuffer));
}

struct SceneData {
    // Mandelbulb:
    float n;
    unsigned int iterationLimit;
    float pseudoInfinity;

    // Colors:
    glm::vec3 skyColor;
    glm::vec3 horizontColor;
    glm::vec3 groundColor;
    glm::vec3 color0;
    glm::vec3 color1;
    glm::vec3 color2;
    float coloringMultiplier;
    float coloringPower;

    // Shading:
    glm::vec3 dirLightDirection;
    glm::vec3 dirLightPower;
    float dirLightIntensity;
    glm::vec3 pointLightPosition;
    glm::vec3 pointLightPower;
    float pointLightIntensity;
    glm::vec3 ambientPower;
    float diffuseIntensity;
    float specularIntensity;
    float ambientIntensity;
    float edgeIntensity;
    float shininess;
    float opacityScale;
    unsigned int shadowStepCount;
    float tintedAttenuationAmount;

    // Fidelity:
    unsigned int rayResolution;
};

SceneData* sceneData;

void allocateAdditionalRenderDataOnDevice()
{
    checkCudaError(hipMalloc(&sceneData, sizeof(SceneData)));
}

void freeAdditionalRenderDataOnDevice()
{
    checkCudaError(hipFree(sceneData));
}

void exportVulkanImageToCuda_R8G8B8A8Unorm(void* mem, VkDeviceSize size, VkDeviceSize offset, uint32_t width, uint32_t height)
{
    imageWidth = width;
    imageHeight = height;

    // import memory into cuda through native handle (win32)
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32; // after win8
    cudaExtMemHandleDesc.handle.win32.handle = mem; // allocation handle
    cudaExtMemHandleDesc.size = size; // allocation size
   
    checkCudaError(hipImportExternalMemory(&cudaExtMemImageBuffer, &cudaExtMemHandleDesc));

    // extract mipmapped array from memory
    cudaExternalMemoryMipmappedArrayDesc externalMemoryMipmappedArrayDesc;
    memset(&externalMemoryMipmappedArrayDesc, 0, sizeof(externalMemoryMipmappedArrayDesc));

    // we want ot interpret the raw memory as an image so we need to specify its format and layout
    hipExtent extent = make_hipExtent(width, height, 0);
    hipChannelFormatDesc formatDesc; // 4 channel, 8 bit per channel, unsigned
    formatDesc.x = 8;
    formatDesc.y = 8;
    formatDesc.z = 8;
    formatDesc.w = 8;
    formatDesc.f = hipChannelFormatKindUnsigned;

    externalMemoryMipmappedArrayDesc.offset = offset; // the image starts here
    externalMemoryMipmappedArrayDesc.formatDesc = formatDesc;
    externalMemoryMipmappedArrayDesc.extent = extent;
    externalMemoryMipmappedArrayDesc.flags = 0;
    externalMemoryMipmappedArrayDesc.numLevels = 1; // no mipmapping
    checkCudaError(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipmappedImageArray, cudaExtMemImageBuffer, &externalMemoryMipmappedArrayDesc));

    // extract first level
    hipArray_t cudaMipLevelArray;
    checkCudaError(hipGetMipmappedArrayLevel(&cudaMipLevelArray, cudaMipmappedImageArray, 0));

    // create surface object for writing
    hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = cudaMipLevelArray;
    
    checkCudaError(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));
}

hipExternalSemaphore_t cudaWaitsForVulkanSemaphore, vulkanWaitsForCudaSemaphore;

void freeExportedSemaphores()
{
    checkCudaError(hipDestroyExternalSemaphore(cudaWaitsForVulkanSemaphore));
    checkCudaError(hipDestroyExternalSemaphore(vulkanWaitsForCudaSemaphore));
}

void exportSemaphoresToCuda(void* cudaWaitsForVulkanSemaphoreHandle, void* vulkanWaitsForCudaSemaphoreHandle) {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.flags = 0;
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

    externalSemaphoreHandleDesc.handle.win32.handle = cudaWaitsForVulkanSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&cudaWaitsForVulkanSemaphore, &externalSemaphoreHandleDesc));

    externalSemaphoreHandleDesc.handle.win32.handle = vulkanWaitsForCudaSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&vulkanWaitsForCudaSemaphore, &externalSemaphoreHandleDesc));
}


// compresses 4 32bit floats into a 32bit uint
__device__ unsigned int rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float map(float x, float fromMin, float fromMax, float toMin, float toMax)
{
    return toMin + (toMax - toMin) * (x - fromMin) / (fromMax - fromMin);
}


/*
    While and Nylander's formula for the "nth power" of the vector v = (x, y, z)
*/
__device__ glm::vec3 vectorPower(glm::vec3 v, float n)
{
    float r = glm::length(v);
    float phi = atan2f(v.y, v.x);
    float theta = atan2f(sqrtf(v.x * v.x + v.y * v.y), r);
    float powRN = powf(r, n);
    return glm::vec3(
                powRN * sinf(n * theta) * cosf(n * phi),
                powRN * sinf(n * theta) * sinf(n * phi),
                powRN * cosf(n * theta)
           );
}


__device__ float mandelbulb(
                        glm::vec3 c,
                        SceneData* scene
)
{
    c = glm::vec3(c.x, c.z, c.y);    // Rearrange coordinates to change the orietation of the system
    
    glm::vec3 v = glm::vec3{ 0.0f, 0.0f, 0.0f };


    for (unsigned int i = 0; i < scene->iterationLimit; i++) {
        v = vectorPower(v, scene->n) + c;
        if (glm::length(v) > scene->pseudoInfinity) {  // divergent iteration
            break;
        }
    }
    return glm::length(v) / scene->pseudoInfinity;   // Outside the object
}


/*
    Code source: https://viclw17.github.io/2018/07/16/raytracing-ray-sphere-intersection
*/
__device__ glm::vec2 hit_sphere(const glm::vec3& center, float radius, const glm::vec3& rayStart, const glm::vec3& rayDir){
    glm::vec3 oc = rayStart - center;
    float a = glm::dot(rayDir, rayDir);
    float b = 2.0f * glm::dot(oc, rayDir);
    float c = glm::dot(oc,oc) - radius * radius;
    float discriminant = b * b - 4 * a * c;
    if(discriminant < 0.0f){
        return glm::vec2{ -1.0f, -1.0f };
    }
    else{
        return glm::vec2{(-b - sqrt(discriminant)) / (2.0*a), (-b + sqrt(discriminant)) / (2.0*a) };
    }
}


__device__ glm::vec4 transferColor(
                        float w,
                        SceneData* scene
                     )
{
    float bias = 0.0f;
    float a = powf(1.0f - w + bias, 0.5f);
    if (w - bias > 1.0f) {
        a = 0.0f;
    }
    float t = powf(fminf(fmaxf(w, 0.0f), 1.0f) * scene->coloringMultiplier, scene->coloringPower);
    if (t < 0.5) {
        return glm::vec4(scene->color0 * (1.0f - t * 2.0f) + scene->color1 * t * 2.0f, a);
    }
    else {
        return glm::vec4(scene->color1 * (1.0f - (t - 0.5f) * 2.0f) + scene->color2 * (t - 0.5f) * 2.0f, a);
    }
}

__device__ glm::vec3 tintedAttenuation(glm::vec4& rgba, float mix)
{
    return rgba.w * ((1.0f - mix) * glm::vec3(1.0f, 1.0f, 1.0f) + mix * glm::vec3( 1.0f - rgba.x, 1.0f - rgba.y, 1.0f - rgba.z ));
}

__device__ glm::vec3 shadingFunction(SceneData* scene, float sourceIntensity, glm::vec3& pos, glm::vec3& albedo, glm::vec3& lightPower, glm::vec3& toLight, glm::vec3& normal, glm::vec3&& viewDir, float shadowStepSize, bool isAmbient)
{
                float lightDistanceSqr = glm::dot(toLight, toLight);
                glm::vec3 lightDir = toLight / sqrtf(lightDistanceSqr);
                glm::vec3 halfway = glm::normalize(lightDir + viewDir);
                glm::vec3 shadow = glm::vec3(0.0f);
                glm::vec3 shadowC = pos + shadowStepSize * lightDir;
                float shadowStepWeight = 1.0f;
                for (unsigned int j = 0; j < scene->shadowStepCount; j++) {
                    float shadowSample = mandelbulb(shadowC, scene);
                    glm::vec4 shadowRGBA = transferColor(shadowSample, scene);
                    glm::vec3 temp = shadowStepSize * scene->opacityScale * shadowStepWeight * tintedAttenuation(shadowRGBA, scene->tintedAttenuationAmount);
                    temp = glm::vec3(fmax(fminf(temp.x, 1.0f), 0.0f), fmaxf(fminf(temp.y, 1.0f), 0.0f), fmaxf(fminf(temp.z, 1.0f), 0.0f));
                    shadow = shadow * (1.0f - temp) + temp;     // Over operator
                    shadowC += shadowStepSize * lightDir;
                    shadowStepWeight *= 0.99f;
                }
                return (powf(fabsf(glm::dot(viewDir, normal)), 3.0f * scene->edgeIntensity) * scene->edgeIntensity + (1.0f - scene->edgeIntensity))  // Edge shading
                    * (
                        + scene->diffuseIntensity * (1.0f - shadow) * sourceIntensity * lightPower / lightDistanceSqr * albedo * fmaxf(glm::dot(lightDir, normal), 0.0f)                         // Diffuse
                        + scene->specularIntensity * (1.0f - shadow) * sourceIntensity * lightPower / lightDistanceSqr * powf(fmaxf(glm::dot(halfway, normal), 0.0f), scene->shininess)   // Specular
                        + ((isAmbient)? scene->ambientIntensity * scene->ambientPower * albedo : glm::vec3(0.0f))                                                                               // Ambient
                    );
}

__global__ void rayCastMandelbulb(hipSurfaceObject_t dstSurface, size_t width, size_t height, glm::vec3 eyePos, glm::mat4 rayDirMtx,                                    
                                    float rotationRad,
                                    SceneData* scene
                                    )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float ndcX = map(float(x), 0.0f, (float)width - 1.0f, -1.0f, 1.0f);
    float ndcY = map(float(y), 0.0f, (float)height - 1.0f, -1.0f, 1.0f);
    
    glm::vec4 rayDir4 = rayDirMtx * glm::vec4(ndcX, ndcY, 0.0f, 1.0f);
    glm::vec3 rayDir = glm::normalize(glm::vec3(rayDir4.x, rayDir4.y, rayDir4.z));

    // rotate coordinate system to rotate the bulb:
    glm::mat4 rotationMat(1.0f);
    rotationMat = glm::rotate(rotationMat, -rotationRad, glm::vec3(0.0, 1.0, 0.0));
    glm::vec4 v4 = rotationMat * glm::vec4(eyePos, 1.0f); 
    eyePos = glm::vec3(v4.x, v4.y, v4.z);
    v4 = rotationMat * glm::vec4(rayDir, 0.0f);
    rayDir = glm::vec3(v4.x, v4.y, v4.z);

    // We know that the object is in the origin of the coordinate system inside a sqrt(2) sphere:
    glm::vec2 boundingSphereHitDistances = hit_sphere(glm::vec3(0.0f, 0.0f, 0.0f), 1.4142f, eyePos, rayDir);
    glm::vec3 backgroundColor = scene->horizontColor * (1.0f - fabsf(rayDir.y)) + scene->skyColor * fmaxf(rayDir.y, 0.0f) + scene->groundColor * -fminf(rayDir.y, 0.0f);   // Create color gradient in background
    if (boundingSphereHitDistances.y < 0.0f) {    // No intersection with the bounding sphere.
        surf2Dwrite(rgbaFloatToInt(float4(backgroundColor.x, backgroundColor.y, backgroundColor.z, 1.0f)), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
        return;
    }
    boundingSphereHitDistances.x = fmaxf(boundingSphereHitDistances.x, 0.0f);   // If inside the sphere;
    float maxDistance = boundingSphereHitDistances.y - boundingSphereHitDistances.x;     
    float stepSize = maxDistance / (float)scene->rayResolution;
    glm::vec3 c = eyePos + boundingSphereHitDistances.x * rayDir;
    
    glm::vec3 accumulatedColor = glm::vec3(0.0f, 0.0f, 0.0f);
    glm::vec3 accumulatedAttenuation = glm::vec3(0.0f, 0.0f, 0.0f);

    float dx = 0.001f;    // Differentiation step

    for (unsigned int step = 0; step < scene->rayResolution; step++) {
        float sample = mandelbulb(c, scene);
        if (sample < 1.0f) {    // Inside the bulb
            // Sample and approximate gradient:
            float sampleDX = mandelbulb(c + dx * glm::vec3(1.0f, 0.0f, 0.0f), scene);
            float sampleDY = mandelbulb(c + dx * glm::vec3(0.0f, 1.0f, 0.0f), scene);
            float sampleDZ = mandelbulb(c + dx * glm::vec3(0.0f, 0.0f, 1.0f), scene);
            float sampleNDX = mandelbulb(c + dx * glm::vec3(-1.0f, 0.0f, 0.0f), scene);
            float sampleNDY = mandelbulb(c + dx * glm::vec3(0.0f, -1.0f, 0.0f), scene);
            float sampleNDZ = mandelbulb(c + dx * glm::vec3(0.0f, 0.0f, -1.0f), scene);
            glm::vec3 grad = glm::vec3{ sampleDX - sampleNDX, sampleDY - sampleNDY, sampleDZ - sampleNDZ } / dx * 0.5f;
            float gLength = glm::length(grad);
            float avgSample = (sample * 1.25f + sampleDX + sampleDY + sampleDZ + sampleNDX + sampleNDY + sampleNDZ) / 7.25f;
            glm::vec4 rgba = transferColor(avgSample, scene);
            glm::vec3 rgb = glm::vec3(rgba.x, rgba.y, rgba.z);
            glm::vec3 tintedA = tintedAttenuation(rgba, scene->tintedAttenuationAmount);
            glm::vec3 shading = rgb;  // If no gradient is available use the base color as "shaded color"
            
            // Shading:
            if (gLength > 0.00001f) {
                glm::vec3 normal = grad / gLength;  // !!! The gradient now is pointing towards the outside of the object. No (-1) !!!
                // Directional light:
                glm::vec4 lDir4 = rotationMat * glm::vec4(scene->dirLightDirection, 0.0f);
                glm::vec3 toLight = glm::vec3(lDir4.x, lDir4.y, lDir4.z);
                shading = shadingFunction(scene, scene->dirLightIntensity, c, rgb, scene->dirLightPower, toLight, normal, -rayDir, stepSize * 2.0f, true);
                // Point light:
                lDir4 = rotationMat * glm::vec4(scene->pointLightPosition, 1.0f);
                toLight = glm::vec3(lDir4.x, lDir4.y, lDir4.z) - c;
                shading += shadingFunction(scene, scene->pointLightIntensity, c, rgb, scene->pointLightPower, toLight, normal, -rayDir, stepSize * 2.0f, false);
            }
            
            // Accumulation:
            glm::vec3 temp = stepSize * scene->opacityScale * rgba.w * shading;
            accumulatedColor += glm::vec3(fmaxf(fminf(temp.x, 1.0f), 0.0f), fmaxf(fminf(temp.y, 1.0f), 0.0f), fmaxf(fminf(temp.z, 1.0f), 0.0f)) * (1.0f - accumulatedAttenuation);  // Color (under operator)

            temp = stepSize * scene->opacityScale * tintedA;
            accumulatedAttenuation += glm::vec3(fmaxf(fminf(temp.x, 1.0f), 0.0f), fmaxf(fminf(temp.y, 1.0f), 0.0f), fmaxf(fminf(temp.z, 1.0f), 0.0f)) * (1.0f - accumulatedAttenuation);   // Attenuation (under operator)
            if (accumulatedAttenuation.x > 0.95f && accumulatedAttenuation.y > 0.95f && accumulatedAttenuation.z > 0.95f) {
                accumulatedAttenuation = glm::vec3(1.0f, 1.0f, 1.0f);
                break;
            }
        }
        c += stepSize * rayDir;     // Step along the ray
    }
    float4 outColor = float4{
                        accumulatedColor.x + backgroundColor.x * (1.0f - accumulatedAttenuation.x),
                        accumulatedColor.y + backgroundColor.y * (1.0f - accumulatedAttenuation.y),
                        accumulatedColor.z + backgroundColor.z * (1.0f - accumulatedAttenuation.z),
                        1.0f
                      };
    surf2Dwrite(rgbaFloatToInt(outColor), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
}


void renderCuda()
{
    if (!theMandelbulbAnimator.getIsHighFidelityHold()) {
        uint32_t nthreads = 16;
        dim3 dimBlock{ nthreads, nthreads };
        dim3 dimGrid{ imageWidth / nthreads + 1, imageHeight / nthreads + 1 };
    
        
        bool isHighFidelity = theMandelbulbAnimator.popIsHighFidelityRender();

        float n = theMandelbulbAnimator.getN();
        unsigned int iterationLimit = (isHighFidelity)? 300 : 8;

        SceneData hostSceneData =  {
            // Mandelbulb:
            theMandelbulbAnimator.getN(),
            iterationLimit,
            theMandelbulbAnimator.getPseudoInfinity(),
            // Colors:
            theMandelbulbAnimator.getSkyColor(),
            theMandelbulbAnimator.getHorizontColor(),
            theMandelbulbAnimator.getGroundColor(),
            theMandelbulbAnimator.getColor0(),
            theMandelbulbAnimator.getColor1(),
            theMandelbulbAnimator.getColor2(),
            theMandelbulbAnimator.getColoringMultiplier(),
            theMandelbulbAnimator.getColoringPower(),
            // Shading:
            theMandelbulbAnimator.getDirLightDirection(),
            theMandelbulbAnimator.getDirLightPower(),
            theMandelbulbAnimator.getDirLightIntensity(),
            theMandelbulbAnimator.getPointLightPosition(),
            theMandelbulbAnimator.getPointLightPower(),
            theMandelbulbAnimator.getPointLightIntensity(),
            theMandelbulbAnimator.getAmbientPower(),
            theMandelbulbAnimator.getDiffuseIntensity(),
            theMandelbulbAnimator.getSpecularIntensity(),
            theMandelbulbAnimator.getAmbientIntensity(),
            theMandelbulbAnimator.getEdgeIntensity(),
            theMandelbulbAnimator.getShininess(),
            theMandelbulbAnimator.getOpacityScale(),
            (unsigned int)((isHighFidelity)? 10 : 5),
            theMandelbulbAnimator.getTintedAttenuationAmount(),
            // Fidelity:
            (unsigned int)((isHighFidelity)? 3000 : 200)
        };

        hipMemcpy(sceneData, &hostSceneData, sizeof(SceneData), hipMemcpyHostToDevice);

        rayCastMandelbulb<<<dimGrid, dimBlock>>>(
            surfaceObject,
            imageWidth,
            imageHeight,
            thePerspectiveCamera.eyePos(),
            thePerspectiveCamera.rayDirMatrix((float)imageWidth / (float)imageHeight),
            theMandelbulbAnimator.getRotation(),
            sceneData
        );
    }

    //checkCudaError(hipGetLastError());
    //checkCudaError(hipDeviceSynchronize()); // not optimal! should be synced with vulkan using semaphores
}