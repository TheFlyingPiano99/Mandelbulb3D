#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "interop.cuh"
#include "stdio.h"
#include <memory>
#include <cuda/std/complex>
#include "../glfwim/input_manager.h"
#include "../camera/perspective_camera.h"
#include "../mandelbulb/mandelbulb_animator.h"

#define checkCudaError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    }
}

uint32_t imageWidth, imageHeight;
hipExternalMemory_t cudaExtMemImageBuffer; // memory handler to the imported memory allocation
hipMipmappedArray_t cudaMipmappedImageArray; // the image interpreted as a mipmapped array
hipSurfaceObject_t surfaceObject; // surface object to the first mip level of the array. Allows write

void freeExportedVulkanImage()
{
    checkCudaError(hipDestroySurfaceObject(surfaceObject));
    checkCudaError(hipFreeMipmappedArray(cudaMipmappedImageArray));
    checkCudaError(hipDestroyExternalMemory(cudaExtMemImageBuffer));
}

void exportVulkanImageToCuda_R8G8B8A8Unorm(void* mem, VkDeviceSize size, VkDeviceSize offset, uint32_t width, uint32_t height)
{
    imageWidth = width;
    imageHeight = height;

    // import memory into cuda through native handle (win32)
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32; // after win8
    cudaExtMemHandleDesc.handle.win32.handle = mem; // allocation handle
    cudaExtMemHandleDesc.size = size; // allocation size
   
    checkCudaError(hipImportExternalMemory(&cudaExtMemImageBuffer, &cudaExtMemHandleDesc));

    // extract mipmapped array from memory
    cudaExternalMemoryMipmappedArrayDesc externalMemoryMipmappedArrayDesc;
    memset(&externalMemoryMipmappedArrayDesc, 0, sizeof(externalMemoryMipmappedArrayDesc));

    // we want ot interpret the raw memory as an image so we need to specify its format and layout
    hipExtent extent = make_hipExtent(width, height, 0);
    hipChannelFormatDesc formatDesc; // 4 channel, 8 bit per channel, unsigned
    formatDesc.x = 8;
    formatDesc.y = 8;
    formatDesc.z = 8;
    formatDesc.w = 8;
    formatDesc.f = hipChannelFormatKindUnsigned;

    externalMemoryMipmappedArrayDesc.offset = offset; // the image starts here
    externalMemoryMipmappedArrayDesc.formatDesc = formatDesc;
    externalMemoryMipmappedArrayDesc.extent = extent;
    externalMemoryMipmappedArrayDesc.flags = 0;
    externalMemoryMipmappedArrayDesc.numLevels = 1; // no mipmapping
    checkCudaError(cudaExternalMemoryGetMappedMipmappedArray(&cudaMipmappedImageArray, cudaExtMemImageBuffer, &externalMemoryMipmappedArrayDesc));

    // extract first level
    hipArray_t cudaMipLevelArray;
    checkCudaError(hipGetMipmappedArrayLevel(&cudaMipLevelArray, cudaMipmappedImageArray, 0));

    // create surface object for writing
    hipResourceDesc resourceDesc;
    memset(&resourceDesc, 0, sizeof(resourceDesc));
    resourceDesc.resType = hipResourceTypeArray;
    resourceDesc.res.array.array = cudaMipLevelArray;
    
    checkCudaError(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));
}

hipExternalSemaphore_t cudaWaitsForVulkanSemaphore, vulkanWaitsForCudaSemaphore;

void freeExportedSemaphores()
{
    checkCudaError(hipDestroyExternalSemaphore(cudaWaitsForVulkanSemaphore));
    checkCudaError(hipDestroyExternalSemaphore(vulkanWaitsForCudaSemaphore));
}

void exportSemaphoresToCuda(void* cudaWaitsForVulkanSemaphoreHandle, void* vulkanWaitsForCudaSemaphoreHandle) {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0, sizeof(externalSemaphoreHandleDesc));
    externalSemaphoreHandleDesc.flags = 0;
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;

    externalSemaphoreHandleDesc.handle.win32.handle = cudaWaitsForVulkanSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&cudaWaitsForVulkanSemaphore, &externalSemaphoreHandleDesc));

    externalSemaphoreHandleDesc.handle.win32.handle = vulkanWaitsForCudaSemaphoreHandle;
    checkCudaError(hipImportExternalSemaphore(&vulkanWaitsForCudaSemaphore, &externalSemaphoreHandleDesc));
}

// compresses 4 32bit floats into a 32bit uint
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
    rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
        ((unsigned int)(rgba.z * 255.0f) << 16) |
        ((unsigned int)(rgba.y * 255.0f) << 8) |
        ((unsigned int)(rgba.x * 255.0f));
}

__device__ float map(float x, float fromMin, float fromMax, float toMin, float toMax) {
    return toMin + (toMax - toMin) * (x - fromMin) / (fromMax - fromMin);
}


/*
    While and Nylander's formula for the "nth power" of the vector v = (x, y, z)
*/
__device__ glm::vec3 vectorPower(glm::vec3 v, float n)
{
    float r = glm::length(v);
    float phi = atan2f(v.y, v.x);
    float theta = atan2f(sqrtf(v.x * v.x + v.y * v.y), r);
    float powRN = powf(r, n);
    return glm::vec3(
                powRN * sinf(n * theta) * cosf(n * phi),
                powRN * sinf(n * theta) * sinf(n * phi),
                powRN * cosf(n * theta)
           );
}


__device__ glm::vec4 mandelbulb(
                        glm::vec3 c,
                        float n,
                        unsigned int iterationLimit,
                        float pseudoInfinity,
                        float coloringMultiplier,
                        float coloringPower,
                        glm::vec3& color0,
                        glm::vec3& color1,
                        glm::vec3& color2
)
{
    c = glm::vec3(c.x, c.z, c.y);    // Rearrange coordinates to change the orietation of the system
    
    glm::vec3 v = glm::vec3{ 0.0f, 0.0f, 0.0f };


    for (unsigned int i = 0; i < iterationLimit; i++) {
        v = vectorPower(v, n) + c;
        if (glm::length(v) > pseudoInfinity) {  // divergent iteration
            return glm::vec4{ 0.0f, 0.0f, 0.0f, 0.0f };   // Outside the object
        }
    }
    float t = powf(glm::length(v) * coloringMultiplier, coloringPower);
    if (t < 0.5) {
        return glm::vec4(color0 * (1.0f - t * 2.0f) + color1 * t * 2.0f, 1.0f);
    }
    else {
        return glm::vec4(color1 * (1.0f - (t - 0.5f) * 2.0f) + color2 * (t - 0.5f) * 2.0f, 1.0f);
    }
}


__global__ void renderToSurface(hipSurfaceObject_t dstSurface, size_t width, size_t height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float wx = map(float(x), 0.0f, (float)width, 0.0f, 1.0f);
    float wy = map(float(y), 0.0f, (float)height, 0.0f, 1.0f);

    float4 dataOut = float4{ wx, wy, wx, 1.0f };

    surf2Dwrite(rgbaFloatToInt(dataOut), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
}



__global__ void renderMandelbrotToSurface(hipSurfaceObject_t dstSurface, size_t width, size_t height, float pos_x, float pos_y, float zoom)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float wx = map(float(x), 0.0f, (float)width, (-2.0f + pos_x) / zoom, (2.0f + pos_x) / zoom);
    float wy = map(float(y), 0.0f, (float)height, (-2.0f + pos_y) / zoom, (2.0f + pos_y) / zoom);
    
    cuda::std::complex<float> c = cuda::std::complex<float>(wx, wy);
    cuda::std::complex<float> z = 0.0f;
    int n = 100;
    double infinity = 200;
    float4 dataOut = float4{ wx, wy, wx, 1.0f };    // Default (For debug)
    dataOut = float4{ 0.0f, 0.0f, 0.0f, 1.0f };
    for (unsigned int escapeTime = 0; escapeTime < n; escapeTime++) {
        z = z*z + c;
        if (abs(z) > infinity) {
            float fractional = logf(logf(abs(z)) / logf(infinity)) / logf(2.0);
            glm::vec3 color = 0.5f + 0.5f * cos(3.0f + (escapeTime - fractional) * 0.15f + glm::vec3(0.0f, 0.6f, 1.0));    // Coloring
            dataOut = float4{ color.x, color.y, color.z, 1.0f };
            break;
        }
    }
    surf2Dwrite(rgbaFloatToInt(dataOut), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
}

__device__ float4 sphere(float3 c, float radius)
{
    if (norm3df(c.x, c.y, c.z) < radius)
    {
        return float4(1.0f, 1.0f, 1.0f, 0.01f);
    }
        return float4(0.0f, 0.0f, 0.0f, 0.0f);    
}


/*
    Code source: https://viclw17.github.io/2018/07/16/raytracing-ray-sphere-intersection
*/
__device__ glm::vec2 hit_sphere(const glm::vec3& center, float radius, const glm::vec3& rayStart, const glm::vec3& rayDir){
    glm::vec3 oc = rayStart - center;
    float a = glm::dot(rayDir, rayDir);
    float b = 2.0f * glm::dot(oc, rayDir);
    float c = glm::dot(oc,oc) - radius * radius;
    float discriminant = b * b - 4 * a * c;
    if(discriminant < 0.0f){
        return glm::vec2{ -1.0f, -1.0f };
    }
    else{
        return glm::vec2{(-b - sqrt(discriminant)) / (2.0*a), (-b + sqrt(discriminant)) / (2.0*a) };
    }
}


__global__ void rayCastMandelbulb(hipSurfaceObject_t dstSurface, size_t width, size_t height, glm::vec3 eyePos, glm::mat4 rayDirMtx,
                                    float n, unsigned int iterationLimit, float pseudoInfinity,
                                    glm::vec3 skyColor,
                                    glm::vec3 horizontColor,
                                    glm::vec3 groundColor,
                                    float coloringMultiplier, float coloringPower,
                                    glm::vec3 color0,
                                    glm::vec3 color1,
                                    glm::vec3 color2
                                    )
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float ndcX = map(float(x), 0.0f, (float)width - 1.0f, -1.0f, 1.0f);
    float ndcY = map(float(y), 0.0f, (float)height - 1.0f, -1.0f, 1.0f);
    
    glm::vec4 rayDir4 = rayDirMtx * glm::vec4(ndcX, ndcY, 0.0f, 1.0f);
    glm::vec3 rayDir = glm::normalize(glm::vec3(rayDir4.x, rayDir4.y, rayDir4.z));

    // We know that the object is in the origin of the coordinate system inside a sqrt(2) sphere:
    glm::vec2 boundingSphereHitDistances = hit_sphere(glm::vec3(0.0f, 0.0f, 0.0f), 1.4142f, eyePos, rayDir);
    glm::vec3 backgroundColor = horizontColor * (1.0f - fabsf(rayDir.y)) + skyColor * fmaxf(rayDir.y, 0.0f) + groundColor * -fminf(rayDir.y, 0.0f);   // Create color gradient in background
    if (boundingSphereHitDistances.y < 0.0f) {    // No intersection with the bounding sphere.
        surf2Dwrite(rgbaFloatToInt(float4(backgroundColor.x, backgroundColor.y, backgroundColor.z, 1.0f)), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
        return;
    }
    boundingSphereHitDistances.x = fmaxf(boundingSphereHitDistances.x, 0.0f);   // If inside the sphere;
    //unsigned int rayResolution = (unsigned int)(200.0f / (1.0f + 0.01f * boundingSphereHitDistances.x));   // Scale the ray cast resolution dynamically based on the distance from the bounding sphere.
    unsigned int rayResolution = 150;
    float maxDistance = boundingSphereHitDistances.y - boundingSphereHitDistances.x;     
    float stepSize = maxDistance / (float)rayResolution;
    glm::vec3 c = eyePos + boundingSphereHitDistances.x * rayDir;
    glm::vec4 accumulated = glm::vec4(0.0f, 0.0f, 0.0f, 0.0f);
    float dx = stepSize;    // Differentiation step
    float opacityScale = 10.0;
    for (unsigned int step = 0; step < rayResolution; step++) {
        glm::vec4 sample = mandelbulb(c, n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
        if (sample.w > 0.001f) {
            // Approximate gradient:
            glm::vec4 sampleDX = mandelbulb(c + dx * glm::vec3(1.0f, 0.0f, 0.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec4 sampleDY = mandelbulb(c + dx * glm::vec3(0.0f, 1.0f, 0.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec4 sampleDZ = mandelbulb(c + dx * glm::vec3(0.0f, 0.0f, 1.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec4 sampleNDX = mandelbulb(c + dx * glm::vec3(-1.0f, 0.0f, 0.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec4 sampleNDY = mandelbulb(c + dx * glm::vec3(0.0f, -1.0f, 0.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec4 sampleNDZ = mandelbulb(c + dx * glm::vec3(0.0f, 0.0f, -1.0f), n, iterationLimit, pseudoInfinity, coloringMultiplier, coloringPower, color0, color1, color2);
            glm::vec3 grad = glm::vec3{ sampleDX.w - sampleNDX.w, sampleDY.w - sampleNDY.w, sampleDZ.w - sampleNDZ.w } / dx * 0.5f;
            float gLength = glm::length(grad);
            float w = (sample.w + sampleDX.w + sampleDY.w + sampleDZ.w + sampleNDX.w + sampleNDY.w + sampleNDZ.w) / 7.0f;
            glm::vec3 color = (
                                glm::vec3(sample.x, sample.y, sample.z)
                              + glm::vec3(sampleDX.x, sampleDX.y, sampleDX.z)
                              + glm::vec3(sampleDY.x, sampleDY.y, sampleDY.z)
                              + glm::vec3(sampleDZ.x, sampleDZ.y, sampleDZ.z)
                              + glm::vec3(sampleNDX.x, sampleNDX.y, sampleNDX.z)
                              + glm::vec3(sampleNDY.x, sampleNDY.y, sampleNDY.z)
                              + glm::vec3(sampleNDZ.x, sampleNDZ.y, sampleNDZ.z)
                              ) / 7.0f;
            float light = 1.0f;
            if (gLength > 0.001f) {
                glm::vec3 normal = - grad / gLength;
                glm::vec3 lightDir = -rayDir;
                light = powf(fmaxf(glm::dot(lightDir, normal), 0.0f), 10.0f) * 0.9f + 0.1f;
            }
            accumulated.x += opacityScale * w * stepSize * color.x * light * (1.0f - accumulated.w);
            accumulated.y += opacityScale * w * stepSize * color.y * light * (1.0f - accumulated.w);
            accumulated.z += opacityScale * w * stepSize * color.z * light * (1.0f - accumulated.w);
            accumulated.w += fminf(opacityScale * stepSize * w, 1.0f) * (1.0f - accumulated.w);   // opacity (under operator)
            if (accumulated.w > 0.95f)
                break;
        }
        c += stepSize * rayDir;     // Step along the ray
    }
    float4 outColor = float4{
                        accumulated.x + backgroundColor.x * (1.0f - accumulated.w),
                        accumulated.y + backgroundColor.y * (1.0f - accumulated.w),
                        accumulated.z + backgroundColor.z * (1.0f - accumulated.w),
                        1.0f
                      };
    surf2Dwrite(rgbaFloatToInt(outColor), dstSurface, x * 4, y); // expects byte coordinates. 1 pixel = 4 byte
}


void renderCuda()
{
    uint32_t nthreads = 32;
    dim3 dimBlock{ nthreads, nthreads };
    dim3 dimGrid{ imageWidth / nthreads + 1, imageHeight / nthreads + 1 };
    
    float n = theMandelbulbAnimator.getN();
    unsigned int iterationLimit = 10;
    float pseudoInfinity = 16.0f;

    rayCastMandelbulb<<<dimGrid, dimBlock>>>(
        surfaceObject,
        imageWidth,
        imageHeight,
        thePerspectiveCamera.eyePos(),
        thePerspectiveCamera.rayDirMatrix((float)imageWidth / (float)imageHeight),
        n,
        iterationLimit,
        pseudoInfinity,
        theMandelbulbAnimator.getSkyColor(),
        theMandelbulbAnimator.getHorizontColor(),
        theMandelbulbAnimator.getGroundColor(),
        theMandelbulbAnimator.getColoringMultiplier(),
        theMandelbulbAnimator.getColoringPower(),
        theMandelbulbAnimator.getColor0(),
        theMandelbulbAnimator.getColor1(),
        theMandelbulbAnimator.getColor2()
    );

    checkCudaError(hipGetLastError());
    //checkCudaError(hipDeviceSynchronize()); // not optimal! should be synced with vulkan using semaphores
}